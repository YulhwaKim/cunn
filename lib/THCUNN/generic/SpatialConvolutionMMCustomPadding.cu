
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialConvolutionMMCustomPadding.cu"
#else

static inline void THNN_(SpatialConvolutionMMCustomPadding_shapeCheck)(
                         THCState *state,
                         THCTensor *input, THCTensor *gradOutput,
                         THCTensor *weight, THCTensor *bias,
                         int kH, int kW, int dH, int dW, int padH, int padW) {
  THArgCheck(kW > 0 && kH > 0, 9,
             "kernel size should be greater than zero, but got kH: %d kW: %d", kH, kW);
  THArgCheck(dW > 0 && dH > 0, 11,
             "stride should be greater than zero, but got dH: %d dW: %d", dH, dW);
  THArgCheck(!bias || THCTensor_(isContiguous)(state, bias), 5,
             "bias tensor has to be contiguous");
  THCUNN_argCheck(state, weight->nDimension == 2 || weight->nDimension == 4, 5, weight,
                  "2D or 4D weight tensor expected, but got: %s");

  if (bias != NULL) {
    THCUNN_check_dim_size(state, bias, 1, 0, weight->size[0]);
  }

  int ndim = input->nDimension;
  int dimf = 0;
  int dimh = 1;
  int dimw = 2;

  if (ndim == 4) {
    dimf++;
    dimh++;
    dimw++;
  }

  THCUNN_argCheck(state, ndim == 3 || ndim == 4, 2, input,
                  "3D or 4D input tensor expected but got: %s");

  long nInputPlane  = weight->size[1] / (kH * kW);
  long inputHeight  = input->size[dimh];
  long inputWidth   = input->size[dimw];
  long nOutputPlane = weight->size[0];
  long outputHeight = (inputHeight + 2*padH - kH) / dH + 1;
  long outputWidth  = (inputWidth + 2*padW - kW) / dW + 1;

  if (outputWidth < 1 || outputHeight < 1)
      THError("Given input size: (%d x %d x %d). "
              "Calculated output size: (%d x %d x %d). Output size is too small",
              nInputPlane,inputHeight,inputWidth,nOutputPlane,outputHeight,outputWidth);

  THCUNN_check_dim_size(state, input, ndim, dimf, nInputPlane);

  if (gradOutput != NULL) {
    THCUNN_check_dim_size(state, gradOutput, ndim, dimf, nOutputPlane);
    THCUNN_check_dim_size(state, gradOutput, ndim, dimh, outputHeight);
    THCUNN_check_dim_size(state, gradOutput, ndim, dimw, outputWidth);
  }
}

void THNN_(SpatialConvolutionMMCustomPadding_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *weight,
           THCTensor *bias,
           THCTensor *columns,
           THCTensor *ones,
           int padValue,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH) {

  THCUNN_assertSameGPU(state, 5, input, output, weight, columns, ones);
  if (bias) {
    THCUNN_assertSameGPU(state, 2, weight, bias);
  }
  THArgCheck(THCTensor_(isContiguous)(state, weight), 4,
             "weight tensor has to be contiguous");

  int freeWeight = 0;

  // Params:
  int nInputPlane = weight->nDimension == 2 ? weight->size[1]/(kH*kW) : weight->size[1];
  int nOutputPlane = weight->size[0];

  if (weight->nDimension == 4) {
    long s1 = weight->size[0];
    long s2 = weight->size[1] * weight->size[2] * weight->size[3];
    weight = THCTensor_(newWithStorage2d)(state, weight->storage, weight->storageOffset, s1, -1, s2, -1);
    freeWeight = 1;
  }

  THNN_(SpatialConvolutionMMCustomPadding_shapeCheck)
       (state, input, NULL, weight, bias, kH, kW, dH, dW, padH, padW);

  input = THCTensor_(newContiguous)(state, input);
  int batch = 1;
  if (input->nDimension == 3) {
    // Force batch
    batch = 0;
    THCTensor_(resize4d)(state, input, 1, input->size[0], input->size[1], input->size[2]);
  }

  long inputWidth   = input->size[3];
  long inputHeight  = input->size[2];
  long outputWidth  = (inputWidth + 2*padW - kW) / dW + 1;
  long outputHeight = (inputHeight + 2*padH - kH) / dH + 1;

  // Batch size + input planes
  long batchSize = input->size[0];

  // Resize output
  THCTensor_(resize4d)(state, output, batchSize, nOutputPlane, outputHeight, outputWidth);

  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nInputPlane*kW*kH, outputHeight*outputWidth);

  // Define a buffer of ones, for bias accumulation
  // Note: this buffer can be shared with other modules, it only ever gets increased,
  // and always contains ones.
  if (ones->nDimension != 2 || ones->size[0]*ones->size[1] < outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCTensor_(resize2d)(state, ones, outputHeight, outputWidth);
    THCTensor_(fill)(state, ones, ScalarConvert<int, real>::to(1));
  }

  // Helpers
  THCTensor *input_n = THCTensor_(new)(state);
  THCTensor *output_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCTensor_(select)(state, input_n, input, 0, elt);
    THCTensor_(select)(state, output_n, output, 0, elt);

    // Do Bias first:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m_ = nOutputPlane;
    long n_ = outputHeight * outputWidth;
    long k_ = 1;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    if (bias) {
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemm(
      #elif defined(THC_REAL_IS_HALF)
      THCudaBlas_Hgemm(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemm(
      #endif
          state,
          't', 'n',
          n_, m_, k_,
          ScalarConvert<int, real>::to(1),
          THCTensor_(data)(state, ones), k_,
          THCTensor_(data)(state, bias), k_,
          ScalarConvert<int, real>::to(0),
          THCTensor_(data)(state, output_n), n_
      );
    } else {
      THCTensor_(zero)(state, output_n);
    }

    // Extract columns:
//     im2col(
//       THCState_getCurrentStream(state),
//       THCTensor_(data)(state, input_n),
//       nInputPlane, inputHeight, inputWidth, kH, kW, padH, padW, dH, dW,
//       1, 1, THCTensor_(data)(state, columns)
//     );
    im2col_custom_padding(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, input_n),
      padValue,
      nInputPlane, inputHeight, inputWidth, kH, kW, padH, padW, dH, dW,
      1, 1, THCTensor_(data)(state, columns)
    );

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m = nOutputPlane;
    long n = columns->size[1];
    long k = nInputPlane*kH*kW;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
        state,
        'n', 'n',
        n, m, k,
        ScalarConvert<int, real>::to(1),
        THCTensor_(data)(state, columns), n,
        THCTensor_(data)(state, weight), k,
        ScalarConvert<int, real>::to(1),
        THCTensor_(data)(state, output_n), n
    );
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, output_n);
  if (freeWeight)
    THCTensor_(free)(state, weight);

  // Resize output
  if (batch == 0) {
    THCTensor_(resize3d)(state, output, nOutputPlane, outputHeight, outputWidth);
    THCTensor_(resize3d)(state, input, nInputPlane, inputHeight, inputWidth);
  }

  THCTensor_(free)(state, input);
}

void THNN_(SpatialConvolutionMMCustomPadding_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *weight,
           THCTensor *gradColumns,
           THCTensor *ones,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH) {

  THCUNN_assertSameGPU(state, 5, input, gradOutput, weight,
                       gradColumns, gradInput);
  THArgCheck(THCTensor_(isContiguous)(state, weight), 4,
             "weight tensor has to be contiguous");

  // Params
  int nInputPlane = weight->nDimension == 2 ? weight->size[1]/(kW*kH) : weight->size[1];
  int nOutputPlane = weight->size[0];

  int freeWeight = 0;
  if (weight->nDimension == 4) {
    long s1 = weight->size[0];
    long s2 = weight->size[1] * weight->size[2] * weight->size[3];
    weight = THCTensor_(newWithStorage2d)(state, weight->storage, weight->storageOffset, s1, -1, s2, -1);
    freeWeight = 1;
  }

  THNN_(SpatialConvolutionMMCustomPadding_shapeCheck)
       (state, input, gradOutput, weight, NULL, kH, kW, dH, dW, padH, padW);

  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  int batch = 1;
  if (input->nDimension == 3) {
    // Force batch
    batch = 0;
    THCTensor_(resize4d)(state, input, 1, input->size[0], input->size[1], input->size[2]);
    THCTensor_(resize4d)(state, gradOutput, 1, gradOutput->size[0], gradOutput->size[1], gradOutput->size[2]);
  }

  long inputWidth   = input->size[3];
  long inputHeight  = input->size[2];
  long outputWidth  = (inputWidth + 2*padW - kW) / dW + 1;
  long outputHeight = (inputHeight + 2*padH - kH) / dH + 1;

  // Batch size + input planes
  long batchSize = input->size[0];

  // Resize output
  THCTensor_(resize4d)(state, gradInput, batchSize, nInputPlane, inputHeight, inputWidth);

  // Resize temporary columns
  THCTensor_(resize2d)(state, gradColumns, nInputPlane*kW*kH, outputHeight*outputWidth);

  // Helpers
  THCTensor *gradInput_n = THCTensor_(new)(state);
  THCTensor *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per sample:
    THCTensor_(select)(state, gradInput_n, gradInput, 0, elt);
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m = nInputPlane*kW*kH;
    long n = gradColumns->size[1];
    long k = nOutputPlane;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
        state,
        'n', 't',
        n, m, k,
        ScalarConvert<int, real>::to(1),
        THCTensor_(data)(state, gradOutput_n), n,
        THCTensor_(data)(state, weight), m,
        ScalarConvert<int, real>::to(0),
        THCTensor_(data)(state, gradColumns), n
    );

    // Unpack columns back into input:
    col2im<real, accreal>(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, gradColumns),
      nInputPlane, inputHeight, inputWidth, outputHeight, outputWidth, kH, kW, padH, padW, dH, dW,
      1, 1, THCTensor_(data)(state, gradInput_n)
    );
  }

  // Free
  THCTensor_(free)(state, gradInput_n);
  THCTensor_(free)(state, gradOutput_n);
  if (freeWeight)
    THCTensor_(free)(state, weight);

  // Resize output
  if (batch == 0) {
    THCTensor_(resize3d)(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
    THCTensor_(resize3d)(state, input, nInputPlane, inputHeight, inputWidth);
    THCTensor_(resize3d)(state, gradInput, nInputPlane, inputHeight, inputWidth);
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
}

void THNN_(SpatialConvolutionMMCustomPadding_accGradParameters)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradWeight,
           THCTensor *gradBias,
           THCTensor *columns,
           THCTensor *ones,
           int padValue,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           accreal scale_) {

  real scale = ScalarConvert<accreal, real>::to(scale_);
  THCUNN_assertSameGPU(state, 5, input, gradOutput, gradWeight, columns, ones);
  if (gradBias) {
   THCUNN_assertSameGPU(state, 2, gradWeight, gradBias);
  }
  THArgCheck(THCTensor_(isContiguous)(state, gradWeight), 4,
             "weight tensor has to be contiguous");

  // Params
  int nInputPlane = gradWeight->nDimension == 2 ? gradWeight->size[1]/(kW*kH) : gradWeight->size[1];
  int nOutputPlane = gradWeight->size[0];

  int freeWeight = 0;
  if (gradWeight->nDimension == 4) {
    long s1 = gradWeight->size[0];
    long s2 = gradWeight->size[1] * gradWeight->size[2] * gradWeight->size[3];
    gradWeight = THCTensor_(newWithStorage2d)(state, gradWeight->storage, gradWeight->storageOffset, s1, -1, s2, -1);
    freeWeight = 1;
  }

  THNN_(SpatialConvolutionMMCustomPadding_shapeCheck)
       (state, input, gradOutput, gradWeight, gradBias, kH, kW, dH, dW, padH, padW);

  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  int batch = 1;
  if (input->nDimension == 3) {
    // Force batch
    batch = 0;
    THCTensor_(resize4d)(state, input, 1, input->size[0], input->size[1], input->size[2]);
    THCTensor_(resize4d)(state, gradOutput, 1, gradOutput->size[0], gradOutput->size[1], gradOutput->size[2]);
  }

  long inputWidth   = input->size[3];
  long inputHeight  = input->size[2];
  long outputWidth  = (inputWidth + 2*padW - kW) / dW + 1;
  long outputHeight = (inputHeight + 2*padH - kH) / dH + 1;

  // Batch size + input planes
  long batchSize = input->size[0];

  // Define a buffer of ones, for bias accumulation
  if (ones->nDimension != 2 || ones->size[0]*ones->size[1] < outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCTensor_(resize2d)(state, ones, outputHeight, outputWidth);
    THCTensor_(fill)(state, ones, ScalarConvert<int, real>::to(1));
  }

  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nInputPlane*kW*kH, outputHeight*outputWidth);

  // Helpers
  THCTensor *input_n = THCTensor_(new)(state);
  THCTensor *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCTensor_(select)(state, input_n, input, 0, elt);
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    // Extract columns:
//     im2col(
//       THCState_getCurrentStream(state),
//       THCTensor_(data)(state, input_n),
//       nInputPlane, inputHeight, inputWidth, kH, kW, padH, padW, dH, dW,
//       1, 1, THCTensor_(data)(state, columns)
//     );
    im2col_custom_padding(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, input_n),
      padValue,
      nInputPlane, inputHeight, inputWidth, kH, kW, padH, padW, dH, dW,
      1, 1, THCTensor_(data)(state, columns)
    );

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m = nOutputPlane;
    long n = nInputPlane*kW*kH;
    long k = columns->size[1];

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
        state,
        't', 'n',
        n, m, k,
        scale,
        THCTensor_(data)(state, columns), k,
        THCTensor_(data)(state, gradOutput_n), k,
        ScalarConvert<int, real>::to(1),
        THCTensor_(data)(state, gradWeight), n
    );

    // Do Bias:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m_ = nOutputPlane;
    long k_ = outputHeight * outputWidth;

    // Do GEMV (note: this is a bit confusing because gemv assumes column-major matrices)
    if (gradBias) {
      #if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemv(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemv(
      #endif
          state,
          't',
          k_, m_,
          scale,
          THCTensor_(data)(state, gradOutput_n), k_,
          THCTensor_(data)(state, ones), 1,
          ScalarConvert<int, real>::to(1),
          THCTensor_(data)(state, gradBias), 1
      );
      #endif
      #ifdef THC_REAL_IS_HALF
      THCudaBlas_Hgemm(
          state,
          't', 'n',
          m_, 1, k_,
          scale,
          THCTensor_(data)(state, gradOutput_n), k_,
          THCTensor_(data)(state, ones), k_,
          ScalarConvert<int, real>::to(1),
          THCTensor_(data)(state, gradBias), m_
      );
      #endif
    }
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, gradOutput_n);
  if (freeWeight)
    THCTensor_(free)(state, gradWeight);

  // Resize
  if (batch == 0) {
    THCTensor_(resize3d)(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
    THCTensor_(resize3d)(state, input, nInputPlane, inputHeight, inputWidth);
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
}

#endif