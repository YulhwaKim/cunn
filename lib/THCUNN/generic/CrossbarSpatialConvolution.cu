
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/CrossbarSpatialConvoluion.cu"
#else

static inline void THNN_(CrossbarSpatialConvolution_shapecheck)(
                         THCState *state,
                         THCTensor *input, THCTensor *weight,
                         int kH, int kW, int dH, int dW, int padH, int padW){
  THArgCheck(kW > 0 && kH > 0, 9,
            "kernel size should be greater than zero, but got kH: %d kW: %d", kH, kW);
  THArgCheck(dW > 0 && dH > 0, 11,
            "stride should be greater than zero, but got dH: %d dW: %d", dH, dW);
  THArgCheck(state, weight->nDimension == 2 || weight->nDimension == 4, 5, weight,
            "2D or 4D weight tensor expected, but got: %s");
  
  int ndim = input->nDimension;
  int dimf = 0;
  int dimh = 1;
  int dimw = 2;
  
  if (ndim == 4) {
    dimf++;
    dimh++;
    dimw++;
  }
  
  THCUNN_argcheck(state, ndim == 3 || ndim == 4, 2, input,
                 "3D or 4D input tensor expected but got: %s");
  
  long nInputPlane = weight->size[1] / (kH * kW);
  long inputHeight = input->size[dimh];
  long inputWidth = input->size[dimw];
  long nOutputPlane = weight->size[0];
  long outputHeight = (inputHeight + 2*padH - kH) / dH + 1;
  long outputWidth = (inputWidth + 2*padW - kW) / dW + 1;
  
  if (outputWidth < 1 || outputHeight < 1)
    THError("Given input size: (%d x %d x %d). "
            "Calculated output size: (%d x %d x %d). Output size is too small", 
             nInputPlane, inputHeight, inputWidth, nOutputPlane, outputHeight, outputWidth);
  
  THCUNN_check_dim_size(state, input, ndim, dimf, nInputPlane); 
}

void THNN_(CrossbarSpatialConvolution_updateoutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *weight,
           THCTensor *columns,
           int accumN,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH) {
  
  THCUNN_assertSameGPU(state, 4, input, output, weight, columns);
  THArgCheck(THCTensor_(isContiguous)(state, weight), 4,
             "weight tensor has to be contiguous");
  
  // convert 4D weight into 2D weight
  int freeWeight = 0;
  if (weight->nDimension == 4) {
    long s1 = weight->size[0];
    long s2 = weight->size[1] * weight->size[2] * weight->size[3];
    weight = THCTensor_(newWithStorage2d)(state, weight->storage, weight->storageOffset, s1, -1, s2, -1);
    freeWeight = 1;
  }
  
  THNN_(CrossbarSpatialConvolution_shapeCheck)
    (sate, input, weight, kH, kW, dH, dW, padH, padW);
  
  // make input contiguous and 4D
  input = THCTensor_(newContiguous)(state, input);
  int batch = 1;
  if (input->nDimension == 3) {
    // Force batch
    batch = 0;
    THCTensor_(resize4d)(state, input, 1, input->size[0], input->size[1], input->size[2]);
  }
  
  // Params:
  long nInputPlane = weight->size[1]/(kH*kW);
  long nIn = weight->size[1];
  long nOutputPlane = weight->size[0];
  long inputWidth = input->size[3];
  long inputHeight = input->size[2];
  long outputWidth = (inputWidth + 2*padW - kW) / dW + 1;
  long outputHeight = (inputHeight + 2*padH - kH) / dH + 1;
  long nOutSpatial = outputWidth * outputHeight;
  long batchSize = input->size[0];
  long nPsum = weight->size[1] / accumN;
  //Check if nPsum is valid
  THArgCheck(nPsum > 0 && weight->size[1] == nPsum * accumN, 101,
            "Number of input per convolution should be divisible by accumN, but we got number of input: %ld, accumN: %d, nPsum: %ld",
             weight->size[1], accumN, nPsum);
  
  // Resize output
  THCTensor_(resize5d)(state, output, batchSize, nOutputPlane, outputHeight, outputWidth, nPsum);
  
  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nInputPlane*kW*kH, outputHeight*outputWidth);
  
  // Helpers
  THCTensor *input_n = THCTensor_(new)(state);
  THCTensor *output_n = THCTensor_(new)(state);
  
  // set dimension of block and grid
  dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid((nOutputPlane+threads.x-1)/threads.x, (nOutSpatial+threads.y-1)/threads.y);
  
  // For each elt in batch, do:
  for (long elt = 0; elt < batchSize; elt ++) {
    // Matrix multiply per output:
    THCTensor_(select)(state, input_n, input, 0, elt);
    THCTensor_(select)(state, output_n, output, 0, elt);
    
    // Extract columns:
    im2col(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, input_n),
      nInputPlane, inputHeight, inputWidth, kH, kW, padH, padW, dH, dW,
      1, 1, THCTensor_(data)(state, columns)
    );
    
    // Execute the kernel
    cunn_CrossbarSpatialConvolution_updateOutput_frame_kernel<real, accreal><<<grid, threads>>>(
          THCTensor_(data)(state, output_n),
          THCTensor_(data)(state, columns),
          THCTensor_(data)(state, weight),
          accumN,
          nIn,
          nOutSpatial,
          nOutputPlane,
          nPsum);
  }
  
  // free memorys
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, output_n);
  if (freeWeight)
    THCTensor_(free)(state, weight);
  
  // Resize output
  if (batch == 0) {
    THCTensor_(resize3d)(state, output, nOutputPlane, outputHeight, outputWidth, nPsum);
    THCTensor_(resize3d)(state, input, nInputPlane, inputHeight, inputWidth);
  }
  THCTensor_(free)(state, input);
  
}


#endif
